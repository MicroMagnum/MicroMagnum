#include "kernels.h"

#include <hip/hip_runtime_api.h>

void configure_transpose_kernels()
{
	static bool init = false;
	if (!init) {
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_rotate_left_3d ), hipFuncCachePreferL1);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_rotate_right_3d), hipFuncCachePreferL1);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_transpose_2d), hipFuncCachePreferShared);
		init = true;
	}
}
