#include "transpose_2d.h"

#include <cassert>

#include "kernels.h"

#include <iostream>
using namespace std;

void ng_cuda_transpose_zeropad_c2c_2d(
	hipStream_t s0, hipStream_t s1,
	int dim_x, int dim_y, // input size
	int exp_y,
	const float *in, // size: dim_x * dim_y
	      float *out // size: exp_y * dim_x (zero-padded in x-direction from dim_y -> exp_y)
)
{
	configure_transpose_kernels();

	const int  in_stride_x = 1;
	const int  in_stride_y = in_stride_x * dim_x;

	const int out_stride_x = 1;
	const int out_stride_y = out_stride_x * exp_y;

	// I. Transpose part
	{
		// Setup grid and block sizes
		const dim3 threads = dim3(16, 16, 1);
		const dim3 grid    = dim3((dim_x-1)/threads.x+1, (dim_y-1)/threads.y+1, 1);

		kernel_transpose_2d<<<grid, threads, 0, s0>>>(dim_x, dim_y, in, in_stride_y, out, out_stride_y);
		checkCudaLastError("kernel_transpose_2d execution failed");
	}

	// II. Zeropad part. 
	if (exp_y > dim_y) {
		const dim3 threads = dim3(16, 16, 1);
		const dim3 grid    = dim3(((exp_y-dim_y)-1)/threads.x+1, (dim_x-1)/threads.y+1, 1);

		kernel_clear_2d<<<grid, threads, 0, s1>>>(exp_y - dim_y, dim_x, out + 2*dim_y, out_stride_y);
		checkCudaLastError("kernel_clear_2d execution failed");
	}

	CUDA_THREAD_SYNCHRONIZE();
}

void ng_cuda_transpose_unpad_c2c_2d(
	hipStream_t s0,
	int dim_x, int dim_y, // input size
	int red_x, // red_x <= dim_x
	const float * in, // size: dim_x * dim_y
	      float *out) // size: dim_y * red_x
{
	configure_transpose_kernels();

	const int  in_stride_x = 1;
	const int  in_stride_y = in_stride_x * dim_x;

	const int out_stride_x = 1;
	const int out_stride_y = out_stride_x * dim_y;

	const dim3 threads = dim3(16, 16, 1);
	const dim3 grid    = dim3((red_x-1)/threads.x+1, (dim_y-1)/threads.y+1, 1);

	kernel_transpose_2d<<<grid, threads, 0, s0>>>(red_x, dim_y, in, in_stride_y, out, out_stride_y);
	checkCudaLastError("kernel_transpose_2d execution failed");

	CUDA_THREAD_SYNCHRONIZE();
}

