#include "hip/hip_runtime.h"
/*
 * Copyright 2012, 2013 by the Micromagnum authors.
 *
 * This file is part of MicroMagnum.
 * 
 * MicroMagnum is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * MicroMagnum is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with MicroMagnum.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "cuda_transpose_zeropad_2d.h"

#include <hip/hip_runtime.h>
#include "matrix/device/cuda_tools.h"

#include <cassert>
//#include <iostream>
//using namespace std;

static const int BLOCK_DIM = 16;

static const bool use_naive_kernel = false;

// This naive transpose kernel suffers from completely non-coalesced writes.
// It can be up to 10x slower than the kernel above for large matrices.
__global__
void kernel_transpose_unpad_c2c_naive(
	int dim_x, int dim_y,
	int red_x,
	const float *in, 
	      float *out)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	// (x,y) run through all output matrix cell positions
	if (x < dim_y && y < red_x) {
		const int  in_idx = 2*(y + dim_x * x);
		const int out_idx = 2*(x + dim_y * y);
		out[out_idx+0] = in[in_idx+0];
		out[out_idx+1] = in[in_idx+1];
	}
}

__global__
void kernel_transpose_unpad_c2c(
	int dim_x, int dim_y,
	int red_x,
	const float *in,
	      float *out)
{
	__shared__ float sh[BLOCK_DIM][BLOCK_DIM*2];

	const int base_x = BLOCK_DIM * blockIdx.x;
	const int base_y = BLOCK_DIM * blockIdx.y;

	const int src_pos_x = base_y+threadIdx.x/2;
	const int src_pos_y = base_x+threadIdx.y;
	if (src_pos_y < dim_y) {
		//     | point to tile @ (0,0) |   |  add row offset  | 
		in  += 2*(base_y + dim_x*base_x) + 2*dim_x*threadIdx.y;

		// copy in-tile to shared mem
		if (src_pos_x             < dim_x) {
			sh[threadIdx.y][threadIdx.x          ] = in[threadIdx.x          ]; // columns 0-7  of row 'threadIdx.y'
		}
		if (src_pos_x+BLOCK_DIM/2 < dim_x) {
			sh[threadIdx.y][threadIdx.x+BLOCK_DIM] = in[threadIdx.x+BLOCK_DIM]; // columns 8-15 of row 'threadIdx.y'
		}
	}

	__syncthreads();

	const int dst_pos_x = base_x+threadIdx.x/2;
	const int dst_pos_y = base_y+threadIdx.y;
	if (dst_pos_y < red_x) {
		//     | point to tile @ (0,0) |   |  add row offset  | 
		out += 2*(base_x + dim_y*base_y) + 2*dim_y*threadIdx.y;

		// copy shared-tile to out-tile in transposed order
		const int imag = threadIdx.x & 1;
		if (dst_pos_x             < dim_y) {
			out[threadIdx.x+        0] = sh[threadIdx.x/2            ][2*threadIdx.y+imag];
		}
		if (dst_pos_x+BLOCK_DIM/2 < dim_y) {
			out[threadIdx.x+BLOCK_DIM] = sh[threadIdx.x/2+BLOCK_DIM/2][2*threadIdx.y+imag];
		}
	}
}

void cuda_transpose_unpad_c2c_2d(
	int dim_x, int dim_y,
	int red_x,
	const float * in_x, const float * in_y, const float * in_z, // size (each): dim_x * dim_y (complex)
	      float *out_x,       float *out_y,       float *out_z) // size (each): dim_y * red_x (complex)
{
	// Setup grid and block sizes
	const dim3 grid((dim_y-1)/BLOCK_DIM+1, (red_x-1)/BLOCK_DIM+1, 1);
	const dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

	const float  *in[3] = { in_x,  in_y,  in_z};
	      float *out[3] = {out_x, out_y, out_z};
	
	for (int c=0; c<3; ++c) {
		if (use_naive_kernel) {
			kernel_transpose_unpad_c2c_naive<<<grid, threads>>>(
				dim_x, dim_y,
				red_x,
				in[c], out[c]
			);
			checkCudaLastError("kernel_transpose_zeropad_c2c_naive execution failed");
		} else {
			kernel_transpose_unpad_c2c<<<grid, threads>>>(
				dim_x, dim_y,
				red_x,
				in[c], out[c]
			);
			checkCudaLastError("kernel_transpose_zeropad_c2c execution failed");
		}
	}
	
	CUDA_THREAD_SYNCHRONIZE();
}

void cuda_transpose_unpad_c2c_2d(
	int dim_x, int dim_y, // input size
	int red_x, // red_x <= dim_x
	const float *in,   // size: dim_x * dim_y
	      float *out)  // size: dim_y * red_x
{
	// Setup grid and block sizes
	const dim3 grid((dim_y-1)/BLOCK_DIM+1, (red_x-1)/BLOCK_DIM+1, 1);
	const dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

	if (use_naive_kernel) {
		kernel_transpose_unpad_c2c_naive<<<grid, threads>>>(
			dim_x, dim_y,
			red_x,
			in, out
		);
		checkCudaLastError("kernel_transpose_zeropad_c2c_naive execution failed");
	} else {
		kernel_transpose_unpad_c2c<<<grid, threads>>>(
			dim_x, dim_y,
			red_x,
			in, out
		);
		checkCudaLastError("kernel_transpose_zeropad_c2c execution failed");
	}
	
	CUDA_THREAD_SYNCHRONIZE();
}
